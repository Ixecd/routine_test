
#include <hip/hip_runtime.h>
#include <cstdio>

/// @brief __global__ 表示让他在GPU上执行, CPU可以通过 xxx<<<1,1>>>调用GPU上的函数
__global__ void kernel() {
    printf("Hello, CUDA!\n");
}

int main() {
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}